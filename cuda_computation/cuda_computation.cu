#include "hip/hip_runtime.h"
﻿//2019 Eric Johnson


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "hip/hip_runtime_api.h"

#include "cuda_computation_common.h"

////////////////////////////////////////////////////////////////////////////////
// GPU-specific defines
////////////////////////////////////////////////////////////////////////////////
//Round a / b to nearest higher integer value
inline int iDivUp(int a, int b)
{
	return (a % b != 0) ? (a / b + 1) : (a / b);
}


__global__ void GPU_DLL_DEMO_kernelCompute(
	int* d_input,
	int* d_output,
	const int adder,
	const int length
)
{
	const int n = (threadIdx.x + blockIdx.x * blockDim.x);

	if (n > length){//handle out of rangeP
		return;
	}

	d_output[n] = d_input[n] + adder;

}

extern "C" void GPU_DLL_DEMO_GPU(
	int* d_input,
	int* d_output,
	const int adder,
	const int length
)
{
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(GPU_DLL_DEMO_kernelCompute), hipFuncCachePreferL1);
	
	dim3 threads(blocksize);
	dim3 blocks(iDivUp(length, threads.x));

	GPU_DLL_DEMO_kernelCompute << <blocks, threads>> >(
		d_input,
		d_output,
		adder,
		length
		);
	getLastCudaError("Kernel() execution failed\n");
}
